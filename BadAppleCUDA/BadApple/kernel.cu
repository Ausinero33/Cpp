#include "hip/hip_runtime.h"
#include "BattleElfEng.h"
#include "Video.h"

#include <opencv2/opencv.hpp>
#include <mmsystem.h>

#include <hip/hip_runtime.h>
#include ""

#define PWIDTH 3
#define PHEIGHT 3

__global__ void processPixel(unsigned char* pixelPtr, CHAR_INFO* screen, int width, int height, int cols, float factor, int cn) {
	int i = threadIdx.x;
	int j = blockIdx.x;

	float meanGS = 0, meanR = 0, meanG = 0, meanB = 0;
	for (int k = 0; k < PHEIGHT; k++) {
		for (int h = 0; h < PWIDTH; h++) {
			meanR += (pixelPtr[(((cols * j * PHEIGHT) + (cols * k)) + (i * PWIDTH + h)) * cn + 2] / factor);
			meanG += (pixelPtr[(((cols * j * PHEIGHT) + (cols * k)) + (i * PWIDTH + h)) * cn + 1] / factor);
			meanB += (pixelPtr[(((cols * j * PHEIGHT) + (cols * k)) + (i * PWIDTH + h)) * cn + 0] / factor);
		}
	}

	short sym;
	short bg_col;
	short fg_col;

	float luminance = 0.2987f * meanR + 0.5870f * meanG + 0.1140f * meanB;
	int pixel_bw = (int)(luminance * 13.0f);
	switch (pixel_bw)
	{
	case 0: bg_col = BG_BLACK; fg_col = FG_BLACK; sym = PIXEL_SOLID; break;

	case 1: bg_col = BG_BLACK; fg_col = FG_DARK_GREY; sym = PIXEL_QUARTER; break;
	case 2: bg_col = BG_BLACK; fg_col = FG_DARK_GREY; sym = PIXEL_HALF; break;
	case 3: bg_col = BG_BLACK; fg_col = FG_DARK_GREY; sym = PIXEL_THREEQUARTERS; break;
	case 4: bg_col = BG_BLACK; fg_col = FG_DARK_GREY; sym = PIXEL_SOLID; break;

	case 5: bg_col = BG_DARK_GREY; fg_col = FG_GREY; sym = PIXEL_QUARTER; break;
	case 6: bg_col = BG_DARK_GREY; fg_col = FG_GREY; sym = PIXEL_HALF; break;
	case 7: bg_col = BG_DARK_GREY; fg_col = FG_GREY; sym = PIXEL_THREEQUARTERS; break;
	case 8: bg_col = BG_DARK_GREY; fg_col = FG_GREY; sym = PIXEL_SOLID; break;

	case 9:  bg_col = BG_GREY; fg_col = FG_WHITE; sym = PIXEL_QUARTER; break;
	case 10: bg_col = BG_GREY; fg_col = FG_WHITE; sym = PIXEL_HALF; break;
	case 11: bg_col = BG_GREY; fg_col = FG_WHITE; sym = PIXEL_THREEQUARTERS; break;
	case 12: bg_col = BG_GREY; fg_col = FG_WHITE; sym = PIXEL_SOLID; break;
	}

	short color = (bg_col | fg_col);

	screen[i + j * width].Char.UnicodeChar = sym;
	screen[i + j * width].Attributes = color;
}

int main() {
	cv::VideoCapture cap(".\\resources\\badapple.mp4");
	short width = (cap.get(cv::CAP_PROP_FRAME_WIDTH) / PWIDTH), height = (cap.get(cv::CAP_PROP_FRAME_HEIGHT) / PHEIGHT);

	PlaySound(TEXT(".\\resources\\badapple.wav"), NULL, SND_FILENAME | SND_ASYNC);

	cv::Mat frame;

	unsigned char* pixelPtr, * dev_pixelPtr;
	int cn;

	float factor = (255.0 * (PHEIGHT * PWIDTH));
	bool flag = true; 

	//WindowCMD wind(width, height, cap.get(cv::CAP_PROP_FPS));
	WindowCMD wind(width, height);

	CHAR_INFO* screen = new CHAR_INFO[width * height];
	CHAR_INFO* dev_screen;

	int a = sizeof(CHAR);

	hipMalloc((void**)&dev_screen, sizeof(CHAR_INFO) * width * height);

	while (true) {
		wind.frameStart = std::chrono::system_clock::now();

		cap >> frame;
		if (!frame.data)
			break;

		pixelPtr = (unsigned char*)frame.data;
		cn = frame.channels();

		if (flag) {
			hipMalloc((void**)&dev_pixelPtr, sizeof(unsigned char) * frame.cols * frame.rows * cn);
			flag = false;
		}

		hipMemcpy(dev_pixelPtr, pixelPtr, sizeof(unsigned char) * frame.cols * frame.rows * cn, hipMemcpyHostToDevice);

		processPixel << <height, width >> > (dev_pixelPtr, dev_screen, width, height, frame.cols, factor, cn);
		//hipDeviceSynchronize();

		hipMemcpy(screen, dev_screen, sizeof(CHAR_INFO) * width * height, hipMemcpyDeviceToHost);

		wind.setScreen(screen);

		wind.draw();
		wind.time->timeFunction();
	}

	cap.release();
}